#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>
#include <string>

#include<math.h>
#include "gpu_hashtable.hpp"

#define PERFORMACE_VIEW 1

#define MIN_LOAD_FACTOR 0.9
#define EMPTY_KEY 	0
#define EMPTY_VALUE 0

#define NUM_BLOCKS(n) (((n) + 255) / 256)
#define NUM_THREADS 256


/* Functie de hash */
__device__ int hash_func(int k, int htable_size)
{
	k = ((k >> 16) ^ k) * 0x45d9f3b;
    k = ((k >> 16) ^ k) * 0x45d9f3b;
	k = (k >> 16) ^ k;
	
	/* rezultatul final trebuie
	 * sa se incadreze in limita data - size
	 */
    return k % htable_size;
}

/* Initializarea hashtable-ului */
__global__ void gpu_init_hashTable(entry_t *htable, const int size)
{
	unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	/* initializarea se face in limitele dimensiunii */
	if (threadId < size) {
		htable[threadId].key = EMPTY_KEY;
		htable[threadId].value = EMPTY_VALUE;
	}
}

/* INIT HASH
 */
GpuHashTable::GpuHashTable(int size) {

	/* alocare de memorie pentru htable */
	hipMalloc(&htable, size * sizeof(entry_t));
	DIE(htable == 0, "hipMalloc htable");

	/* alocare de memorie pentru contorul elementelor din hashtable */
	hipMallocManaged(&count, sizeof(unsigned int));
	DIE(count == 0, "hipMallocManaged count");

	/* initializarea valorilor din hashtable */
	gpu_init_hashTable<<<NUM_BLOCKS(size), NUM_THREADS>>>(htable, size);
	hipDeviceSynchronize();

	/* initializarea marimii hashtable-ului */
	htable_size = size;
	/* initializarea contorului de elemente din hashtable */
	*count = 0;
}

/* DESTROY HASH
 */
GpuHashTable::~GpuHashTable() {
	/* eliberarea memoriei pentru hashtable */
	if (htable != 0)
		hipFree(htable);

	/* eliberarea memoriei pentru contor */
	if (count != 0)
		hipFree(count);
}

/* RESHAPE HASH
 */
void GpuHashTable::reshape(int numBucketsReshape) {
	int size = numBucketsReshape;

	if (htable != 0)
		hipFree(htable);
	
	/* alocare de memorie pentru htable */
	hipMalloc(&htable, size * sizeof(entry_t));
	DIE(htable == 0, "hipMalloc htable");

	if (count != 0)
		hipFree(count);
	
	/* alocare de memorie pentru contorul elementelor din hashtable */
	hipMallocManaged(&count, sizeof(unsigned int));
	DIE(count == 0, "hipMallocManaged count");

	/* initializarea valorilor din hashtable */
	gpu_init_hashTable<<<NUM_BLOCKS(size), NUM_THREADS>>>(htable, size);
	hipDeviceSynchronize();

	/* initializarea marimii hashtable-ului */
	htable_size = size;
	/* initializarea contorului de elemente din hashtable */
	*count = 0;
}

/* Construirea unui hashtable pe baza unui alt hashtable */
__global__ void gpu_hashtable_copy(entry_t *old_htable, entry_t *new_htable, const int old_htable_size, const int new_htable_size)
{
	unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	/* vream sa adaugam elemente care
	 * se gasesc in limitele vechiului hashtable
	 */
	if (threadId >= old_htable_size)
		return;

	/* cheia pentru thread-ul curent din vechiul hashtable */
	int key = old_htable[threadId].key;

	/* pentru perechile goale
	 * nu avem ce aduaga in noul hashtable
	 */
	if (key == EMPTY_KEY)
		return;
		
	int current_key;
	int index = hash_func(key, new_htable_size);	
	
	/* cautarea unui slot valabil pentru adaugarea unei noi perechi cheie:valoare */
	while (1) {
		/* verificam (si actualizam) atomic faptul ca slot-ul este liber */
		current_key = atomicCAS(&new_htable[index].key, EMPTY_KEY, key);
		
		/* daca a fost gasit un slot liber este folosit acesta */
		if (current_key == EMPTY_KEY || current_key == key) {
			new_htable[index].value = old_htable[threadId].value;
			return;
		}

		/* daca slot-ul curent este ocupat cautam in continuare */
		index = (index + 1) % new_htable_size;
	}
}

/* Adugarea unei noi perechi in hashtable */
__global__ void gpu_hashtable_insert(entry_t *htable, unsigned int *count, const int htable_size, const int *keys, const int *values, const int numKeys)
{
	unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	/* id-ul trebuie sa se afle in limitele date */
	if (threadId >= numKeys)
		return;

	int key = keys[threadId];
	int value = values[threadId];
	int current_key;
	int index = hash_func(key, htable_size);

	/* pentru cheia curenta cautam un slot liber pentru a o adauga */
	while (1) {
		/* verificam (si actualizam) atomic faptul ca slot-ul este liber */
		current_key = atomicCAS(&htable[index].key, EMPTY_KEY, key);

		/* daca a fost gasit un slot liber este folosit acesta */
		if (current_key == EMPTY_KEY || current_key == key) {
			htable[index].value = value;
			
			/* daca spatiul era liber inseamna ca a fost adaugat un nou element 
			 * alternativ, aceasta inserare ar fi presupus actualizarea unei valori
			 * caz in care nu ar creste numarul de elemente din hashmap
			 */
			if (current_key == EMPTY_KEY)
				atomicAdd(count, 1);
			return;
		}

		/* daca slot-ul curent este ocupat cautam in continuare */
		index = (index + 1) % htable_size;
	}
}

/* INSERT BATCH
 */
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {
	
	int *device_keys;
	int *device_values;
	int old_htable_size;
	entry_t *new_htable;

	/* verificam faptul ca avem suficient loc pentru noile elemente */
	if (*count + numKeys > MIN_LOAD_FACTOR * htable_size) {

		/* //////////////////////////////////////////////////////////////
		 * ///////////// Redimensionarea hashthable-ului  ///////////////
		 * //////////////////////////////////////////////////////////////
		 */

		old_htable_size = htable_size;
		/* noua marime se caluleaza in functie de gradul de ocupare dorit */
		htable_size = (*count + numKeys) / MIN_LOAD_FACTOR;
		
		/* alocarea memoriei pentru noul hashtable */
		hipMalloc(&new_htable, htable_size * sizeof(entry_t));
		DIE(new_htable == 0, "hipMalloc new_htable");

		/* initializarea noului hashtable */
		gpu_init_hashTable<<<NUM_BLOCKS(htable_size), NUM_THREADS>>>(new_htable, htable_size);
		hipDeviceSynchronize();

		/* introducerea datelor existente in noul hashtable */
		gpu_hashtable_copy<<<NUM_BLOCKS(htable_size), NUM_THREADS>>>(htable, new_htable, old_htable_size, htable_size);
		hipDeviceSynchronize();

		/* eliberarea memoriei vechiului hashtable */
		hipFree(htable);

		/* actualizarea noului hashtable */
		htable = new_htable;
	}

	/* alocare de memorie pentru parametrii pentru kernel */
	hipMalloc((void **) &device_keys, numKeys * sizeof(int));	
	DIE(device_keys == 0, "hipMalloc device_keys");

	hipMalloc((void **) &device_values, numKeys * sizeof(int));
	DIE(device_values == 0, "hipMalloc device_keys");
	
	/* copierea datelor pentru chei si valori */
	hipMemcpy(device_keys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_values, values, numKeys * sizeof(int), hipMemcpyHostToDevice);

#if PERFORMACE_VIEW
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
#endif

	/* inserarea noilor elemente in hashtable */
	gpu_hashtable_insert<<<NUM_BLOCKS(numKeys), NUM_THREADS>>>(htable, count, htable_size, device_keys, device_values, numKeys);
	hipDeviceSynchronize();

#if PERFORMACE_VIEW
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float time = 0;
	hipEventElapsedTime(&time, start, stop);
	float seconds = time / 1000.0f;
	printf("Inserted %d elements in %f ms (%f million keys/second)\n", numKeys, time, numKeys / (double)seconds / 1000000.0f);
#endif

	/* eliberarea memoriei pentru parametrii kernel-ului */
	hipFree(device_keys);
	hipFree(device_values);

	return true;
}

/* Returnarea valorilor pentru un set de chei */
__global__ void gpu_hashtable_lookup(entry_t *htable, const int htable_size, const int *keys, int *values, const int numKeys)
{
	unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	/* id-ul trebuie sa se afle in limitele date */
	if (threadId >= numKeys)
		return;

	int key; 
	int index;
	int timeout = 0;
	key = keys[threadId];
	index = hash_func(key, htable_size);

	/* se cauta valoarea asociata cheii date */
	while (true) {

		/* daca a fost parcurs intreg hashtable-ul
		 * atunci cheia data nu se regaseste in acesta
		 */
		if (timeout == htable_size) {
			values[threadId] = EMPTY_VALUE;
			return;
		}

		/* daca a fost gasita o potrivire
		 * atunci intoarcem valoarea de la acel index
		 */
		if (htable[index].key == key) {
			values[threadId] = htable[index].value;
			return;
		}

		/* daca index-ul curent nu avea cheia cautata 
		 * atunci continuam cautarea
		 */
		index = (index + 1) % htable_size;
		timeout += 1;
	}
}

/* GET BATCH
 */
int* GpuHashTable::getBatch(int* keys, int numKeys) {
	int *values;
	int *device_keys;
	int *device_values;

	/* alocare de memorie pentru a retine valorile cautate */
	values = (int *)malloc(numKeys * sizeof(int));

	/* alocare de memorie pentru parametrii functiei de cautare */
	hipMalloc((void **) &device_keys, numKeys * sizeof(int));	
	DIE(device_keys == 0, "hipMalloc device_keys");

	hipMalloc((void **) &device_values, numKeys * sizeof(int));
	DIE(device_values == 0, "hipMalloc device_keys");

	/* copierea datelor pentru cheile cautate */
	hipMemcpy(device_keys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);

#if PERFORMACE_VIEW
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
#endif

	/* cautarea valorilor asociate cheilor date */
	gpu_hashtable_lookup<<<NUM_BLOCKS(numKeys), NUM_THREADS>>>(htable, htable_size, device_keys, device_values, numKeys);
	hipDeviceSynchronize();

#if PERFORMACE_VIEW
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float time = 0;
	hipEventElapsedTime(&time, start, stop);
	float seconds = time / 1000.0f;
	printf("Got %d elements in %f ms (%f million keys/second)\n", numKeys, time, numKeys / (double)seconds / 1000000.0f);
#endif

	/* copierea valorilor gasite pe Host */
	hipMemcpy(values, device_values, numKeys * sizeof(int), hipMemcpyDeviceToHost);

	return values;
}

/* GET LOAD FACTOR
 * num elements / hash total slots elements
 */
float GpuHashTable::loadFactor() {
	/*
	 * count - numarul de elemente aflate in hashtable
	 * htable_size - spatiul total alocat pentru hashtable
	 */
	return (float)*count /(float)htable_size; // no larger than 1.0f = 100%
}

/*********************************************************/

#define HASH_INIT GpuHashTable GpuHashTable(1);
#define HASH_RESERVE(size) GpuHashTable.reshape(size);

#define HASH_BATCH_INSERT(keys, values, numKeys) GpuHashTable.insertBatch(keys, values, numKeys)
#define HASH_BATCH_GET(keys, numKeys) GpuHashTable.getBatch(keys, numKeys)

#define HASH_LOAD_FACTOR GpuHashTable.loadFactor()

#include "test_map.cpp"
